#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2015 Open Source Robotics Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */
#include <optix.h>
#include <optix_math.h>
#include <gz/rendering/optix/OptixRayTypes.hh>

// camera variables
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3,   u, , );
rtDeclareVariable(float3,   v, , );
rtDeclareVariable(float3,   w, , );
rtDeclareVariable(uint,    aa, , );
rtBuffer<float3, 2> buffer;

// current ray variables
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );

// scene variables
rtDeclareVariable(rtObject, rootGroup, , );
rtDeclareVariable(float, sceneEpsilon, , );

typedef struct Context_t
{

  uint2 subpixel;
  float subpixelWeight;

} Context;

static __inline__ __device__ void AddColor(const uint2 &_index,
    const float3 &_color)
{
  float *channels = (float *)&buffer[_index];
  atomicAdd(&channels[0], _color.x);
  atomicAdd(&channels[1], _color.y);
  atomicAdd(&channels[2], _color.z);
}

static __inline__ __device__ void AddColor(const Context &_context,
    const float3 &_color)
{
  // TODO: clean up

  float3 weightedColor = _color * _context.subpixelWeight;
  AddColor(launchIndex, weightedColor);

  bool addX = _context.subpixel.x == aa - 1 && launchIndex.x < launchDim.x;
  bool addY = _context.subpixel.y == aa - 1 && launchIndex.y < launchDim.y;

  if (addX)
  {
    uint2 index = launchIndex + make_uint2(1, 0);
    AddColor(index, weightedColor);
  }

  if (addY)
  {
    uint2 index = launchIndex + make_uint2(0, 1);
    AddColor(index, weightedColor);
  }

  if (addX && addY)
  {
    uint2 index = launchIndex + make_uint2(1, 1);
    AddColor(index, weightedColor);
  }
}

static __inline__ __device__ void TraceRay(const Context &_context)
{
  float2 offset = make_float2(_context.subpixel) / aa;

  // get image plane intersect point
  float2 pixel = make_float2(launchIndex) + offset;
  float2 size  = make_float2(launchDim);
  float2 ratio = pixel / size - 0.5;

  // create ray that traverses through image plane point
  float3 direction = normalize(ratio.x * u + ratio.y * v + w);
  optix::Ray ray(eye, direction, RT_RADIANCE, sceneEpsilon);

  // initialize ray payload
  OptixRadianceRayData data;
  data.color = make_float3(0, 0, 0);
  data.importance = 1;
  data.depth = 0;

  // trace ray and update buffer
  rtTrace(rootGroup, ray, data);
  AddColor(_context, data.color);
}

static __inline__ __device__ void RenderAA()
{
  Context context;
  context.subpixelWeight = 1.0 / (aa * aa);
  uint &x = context.subpixel.x;
  uint &y = context.subpixel.y;

  for (x = 1; x < aa; ++x)
  {
    for (y = 1; y < aa; ++y)
    {
      TraceRay(context);
    }
  }
}

static __inline__ __device__ void RenderNoAA()
{
  // get image plane intersect point
  float2 pixel = make_float2(launchIndex) + 0.5;
  float2 size  = make_float2(launchDim);
  float2 ratio = pixel / size - 0.5;

  // create ray that traverses through image plane point
  float3 direction = normalize(ratio.x * u + ratio.y * v + w);
  optix::Ray ray(eye, direction, RT_RADIANCE, sceneEpsilon);

  // initialize ray payload
  OptixRadianceRayData data;
  data.color = make_float3(0, 0, 0);
  data.importance = 1;
  data.depth = 0;

  // trace ray and update buffer
  rtTrace(rootGroup, ray, data);
  buffer[launchIndex] = data.color;
}

RT_PROGRAM void Render()
{
  if (aa > 1)
  {
    RenderAA();
  }
  else
  {
    RenderNoAA();
  }
}

RT_PROGRAM void Clear()
{
  buffer[launchIndex] = make_float3(0);
}
