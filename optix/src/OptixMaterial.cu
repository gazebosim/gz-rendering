#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2015 Open Source Robotics Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 *
 */
#include <optix.h>
#include <optix_math.h>
#include <optixu/optixu_matrix_namespace.h>
#include "gz/rendering/optix/OptixRayTypes.hh"
#include "gz/rendering/optix/OptixLightTypes.hh"

// scene variables
rtDeclareVariable(float3, ambientLightColor, , );
rtDeclareVariable(int, maxReflectionDepth, , );
rtDeclareVariable(int, maxRefractionDepth, , );
rtDeclareVariable(float, importanceCutoff, , );
rtDeclareVariable(float, sceneEpsilon, , );
rtDeclareVariable(rtObject, rootGroup, , );
rtBuffer<OptixDirectionalLightData> directionalLights;
rtBuffer<OptixPointLightData> pointLights;
rtTextureSampler<float4, 2> texSampler;
rtTextureSampler<float4, 2> normSampler;
rtDeclareVariable(bool, normWorldSpace, , );

// material variables
rtDeclareVariable(float3, ambient, , );
rtDeclareVariable(float3, diffuse, , );
rtDeclareVariable(float3, emissive, , );
rtDeclareVariable(float, reflectivity, , );
rtDeclareVariable(float, transparency, , );
rtDeclareVariable(uint, lightingEnabled, , );
rtDeclareVariable(uint, castShadows, , );
rtDeclareVariable(uint, receiveShadows, , );

// ray variables
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(OptixRadianceRayData, radianceData, rtPayload, );
rtDeclareVariable(OptixShadowRayData, shadowData, rtPayload, );

// intersect variables
rtDeclareVariable(float, hitDist, rtIntersectionDistance, );
rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );
rtDeclareVariable(float3, shadingTangent, attribute shadingTangent, );
rtDeclareVariable(float2, texCoord, attribute texCoord, );

static __device__ __inline__ float3 Exp(const float3 &_x)
{
  return make_float3(exp(_x.x), exp(_x.y), exp(_x.z));
}

RT_PROGRAM void AnyHit()
{
  float3 shadowAtten   = diffuse;

  if (!castShadows)
  {
    rtIgnoreIntersection();
  }
  else if (transparency > 0)
  {
    float3 worldNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD,
          shadingNormal));

    float ndi = fabs(dot(worldNormal, ray.direction));

    shadowData.attenuation *= 1.0 - fresnel_schlick(ndi, 5, 1 - shadowAtten,
        make_float3(1));

    // float3 ones = make_float3(1);
    // shadowData.attenuation *= 1 - transparency * (ones - ambient) *
    //     (2 - fresnel_schlick(ndi, 5, 1 - shadowAtten, make_float3(1)));

    rtIgnoreIntersection();
  }
  else
  {
    shadowData.attenuation = make_float3(0);
    rtTerminateRay();
  }
}

RT_PROGRAM void ClosestHit()
{
  float  fresnelExp    = 3.0;
  float  fresnelMin    = 0.1;
  float  fresnelMax    = 1.0;
  float  refractIndex  = 1.4;
  float3 refractColor  = diffuse;
  float3 extinctConst  = diffuse;
  float3 cutoffColor   = diffuse;

  float3 color = ambient;
  if (lightingEnabled) color *= ambientLightColor;

  if (emissive.x > 0 || emissive.y > 0 || emissive.z > 0)
    color = emissive;

  float3 worldGeomNorm = normalize(
      rtTransformNormal(RT_OBJECT_TO_WORLD, geometricNormal));

  float3 worldShadeNorm = normalize(
      rtTransformNormal(RT_OBJECT_TO_WORLD, shadingNormal));

  float3 worldShadeTang = normalize(
      rtTransformNormal(RT_OBJECT_TO_WORLD, shadingTangent));

  float3 matNorm = make_float3(tex2D(normSampler, texCoord.x, texCoord.y));

  if ((matNorm.x > 0 || matNorm.y > 0 || matNorm.z > 0) &&
      (shadingTangent.x != 0 || shadingTangent.y != 0 || shadingTangent.z != 0))
  {
    optix::Matrix<3, 3> matrix;
    matrix.setCol(0, worldShadeNorm);
    matrix.setCol(1, worldShadeTang);
    matrix.setCol(2, cross(worldShadeNorm, worldShadeTang));

    matNorm = normalize(matNorm - 0.5);
    worldShadeNorm = matrix * matNorm;
  }

  float3 forwardNormal = faceforward(worldShadeNorm, -ray.direction,
      worldGeomNorm);

  float3 hitPoint = ray.origin + hitDist * ray.direction;

  float reflection = 1.0;
  float3 result = make_float3(0);
  float3 beerAtten = make_float3(1);

  if (transparency > 0)
  {
    float3 beerAtten = (dot(worldShadeNorm, ray.direction) > 0) ?
      Exp(extinctConst * hitDist) : make_float3(1);

    if (radianceData.depth < maxRefractionDepth)
    {
      float3 t;

      if (refract(t, ray.direction, worldShadeNorm, refractIndex))
      {
        float cosTheta = dot(ray.direction, worldGeomNorm);
        cosTheta = (cosTheta < 0) ? -cosTheta : dot(t, worldShadeNorm);

        reflection = fresnel_schlick(cosTheta, fresnelExp, fresnelMin,
            fresnelMax);

        float importance = radianceData.importance * (1 - reflection) *
          optix::luminance(refractColor * beerAtten);

        if (importance > importanceCutoff)
        {
          optix::Ray ray(hitPoint, t, RT_RADIANCE, sceneEpsilon);
          OptixRadianceRayData refrData;
          refrData.depth = radianceData.depth + 1;
          refrData.importance = importance;
          rtTrace(rootGroup, ray, refrData);
          result += (1 - reflectivity) * refractColor * refrData.color;
        }
        else
        {
          result += (1 - reflectivity) * refractColor * cutoffColor;
        }
      }
    }
  }

  // TODO: clean up
  for (int i = 0; i < directionalLights.size() && lightingEnabled; ++i)
  {
    OptixDirectionalLightData light = directionalLights[i];
    float3 l = normalize(-light.direction);
    float ndl = dot(forwardNormal, l);

    if (ndl > 0)
    {
      OptixShadowRayData data;
      data.attenuation = make_float3(1);
      optix::Ray shadowRay(hitPoint, l, RT_SHADOW, sceneEpsilon, 1000);
      rtTrace(rootGroup, shadowRay, data);
      float3 attenuation = data.attenuation;

      if (fmaxf(attenuation) > 0)
      {
        OptixLightAttenuation att = light.common.atten;
        float attp = 1 - fminf(hitDist, att.range) / att.range;

        float attf = att.constant + attp * att.linear + attp *
          attp * att.quadratic;

        // rtPrintf("1 - (%f / %f) = %f\n", hitDist, att.range, attp);

        // rtPrintf("%f + %f * %f + %f * %f * %f = %f\n", att.constant, attp,
        //     att.linear, attp, attp, att.quadratic, attf);

        float4 ld4 = light.common.color.diffuse;
        float3 Lc = make_float3(ld4.x, ld4.y, ld4.z) * attenuation;
        color += diffuse * ndl * Lc * attf;

        float3 H = normalize(l - ray.direction);
        float nDh = dot( forwardNormal, H );

        if(nDh > 0)
        {
          // TODO: include material specular
          // float4 ks4 = light.common.color.specular;
          // float3 Ks = make_float3(ks4.x, ks4.y, ks4.z) * attenuation;
          float3 Ks = make_float3(0.5, 0.5, 0.5);
          float phong_exp = 50;
          color += Ks * Lc * pow(nDh, phong_exp);
        }
      }
    }
  }

  for (int i = 0; i < pointLights.size() && lightingEnabled; ++i)
  {
    OptixPointLightData light = pointLights[i];
    float3 l = normalize(light.common.position - hitPoint);
    float ndl = dot(forwardNormal, l);

    if (ndl > 0)
    {
      OptixShadowRayData data;
      data.attenuation = make_float3(1);
      float dist = length(light.common.position - hitPoint);
      optix::Ray shadowRay(hitPoint, l, RT_SHADOW, sceneEpsilon, dist);
      rtTrace(rootGroup, shadowRay, data);
      float3 attenuation = data.attenuation;

      if (fmaxf(attenuation) > 0)
      {
        // TODO: add light's attenuation
        float4 ld4 = light.common.color.diffuse;
        float3 Lc = make_float3(ld4.x, ld4.y, ld4.z) * attenuation;
        color += diffuse * ndl * Lc;

        float3 H = normalize(l - ray.direction);
        float nDh = dot( forwardNormal, H );

        if(nDh > 0)
        {
          // TODO: include material specular
          // float4 ks4 = light.common.color.specular;
          // float3 Ks = make_float3(ks4.x, ks4.y, ks4.z) * attenuation;
          float3 Ks = make_float3(0.5, 0.5, 0.5);
          float phong_exp = 50;
          color += Ks * Lc * pow(nDh, phong_exp);
        }
      }
    }
  }

  if (reflectivity > 0 && radianceData.depth < maxReflectionDepth)
  {
    OptixRadianceRayData refData;
    refData.depth = radianceData.depth + 1;
    float3 R = reflect(ray.direction, forwardNormal);
    optix::Ray refRay(hitPoint, R, RT_RADIANCE, sceneEpsilon);
    rtTrace(rootGroup, refRay, refData);

    // TODO: determine the actual root of the problem
    if (refData.color.x < 1 || refData.color.y < 1 || refData.color.z < 1)
      color += reflectivity * refData.color;
  }

  float3 tcolor = make_float3(tex2D(texSampler, texCoord.x, texCoord.y));
  float3 finalColor = color + color * tcolor * tcolor * tcolor;

  radianceData.color = (1 - transparency) * finalColor +
      (transparency * result * beerAtten);
}
